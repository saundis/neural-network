#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

// A simple CUDA kernel that runs on the GPU.
__global__ void addKernel(const int *in, int *out, std::size_t n) {
    std::size_t index{ static_cast<std::size_t>(blockIdx.x * blockDim.x + threadIdx.x) };
    if (index < n) {
        out[index] = in[index] + 1;
    }
}

// Manages the memory on the device and launches the kernel.
extern "C" void runCudaKernel(const int* input, int* output, std::size_t n) {
    int* d_in{}, int* d_out{};

    // Allocate memory on the GPU
    hipMalloc(&d_in, n * sizeof(int));
    hipMalloc(&d_out, n * sizeof(int));

    // Copy input data from host to device
    hipMemcpy(d_in, input, n * sizeof(int), hipMemcpyHostToDevice);

    // Define grid and block dimensions for the kernel
    int threadsPerBlock{256};
    int blocksPerGrid{(static_cast<int>(n) + threadsPerBlock - 1) / threadsPerBlock};

    // Launch the kernel
    addKernel<<<blocksPerGrid, threadsPerBlock>>>(d_in, d_out, n);

    hipGetLastError();
    hipDeviceSynchronize();

    hipMemcpy(output, d_out, n * sizeof(int), hipMemcpyDeviceToHost);

    // Clean up: free GPU memory
    hipFree(d_in);
    hipFree(d_out);
}
